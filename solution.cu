#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#include "kernel.cuh"

#define NUM_ELEMENTS 4096

#define SEED_MAX 0.9
#define SEED_MIN 0.7

#define BLOCK_SIZE 1024
#define GRID_SIZE NUM_ELEMENTS/BLOCK_SIZE

extern __constant__ float seed_device_constant[1024];

void random_generator(float* random_bins, float min, float max) {
    srand((unsigned int)time(NULL));
    
    for (int i = 0; i < NUM_ELEMENTS; i++) {
        random_bins[i] = min + ((float)rand() / RAND_MAX) * (max - min);
    }
}

void print_fractal(float* bins) {
    for (int i = 0; i < NUM_ELEMENTS; i++) {
        printf("%d: %.4f\n ", i, bins[i]);
    }
    printf("\n");
}

int main(void) {
    // timing related things
    hipEvent_t astartEvent, astopEvent;
    float aelapsedTime;
    hipEventCreate(&astartEvent);
    hipEventCreate(&astopEvent);

    // initialize arrays
    float* seed_host;
    float* seed_device;

    float* bins_host;
    float* bins_device;

    // generate random seeds
    seed_host = (float*)malloc(NUM_ELEMENTS * sizeof(float));
    random_generator(seed_host, SEED_MIN, SEED_MAX);

    // allocate memory and initialize host_bins
    bins_host = (float*)malloc(NUM_ELEMENTS * sizeof(float));
	for (int i = 0; i < NUM_ELEMENTS; i++) {
		bins_host[i] = 0;
	}
    
    // memcpy the seeds to GPU and malloc the bins on device
    if (hipMalloc((void **)&bins_device, NUM_ELEMENTS * sizeof(float)) != hipSuccess) {
        printf("bins malloc error\n");
    }

    if (hipMalloc((void **)&seed_device, NUM_ELEMENTS * sizeof(float)) != hipSuccess) {
        printf("seed malloc error\n");
    }

    if (hipMemcpy(seed_device, seed_host, NUM_ELEMENTS * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
        printf("seed memcpy error from host to device\n");
    }
	
	if (hipMemcpy(bins_device, bins_host, NUM_ELEMENTS * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
        printf("bins memcpy error from host to device\n");
    }

    dim3 blockDim(BLOCK_SIZE), gridDim(GRID_SIZE);

    // copy constant memory to GPU for optimized
    hipMemcpyToSymbol(HIP_SYMBOL(seed_device_constant), &seed_host, 1024 * sizeof(float)); // seed device needs to be 1024 size

    hipEventRecord(astartEvent, 0);
    
    //volatility_naive<<<blockDim, gridDim>>>(50, bins_device, seed_device);

    volatility_optimized<<<blockDim, gridDim>>>(50, bins_device);

    hipEventRecord(astopEvent, 0);
    hipEventSynchronize(astopEvent);
    hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);
    
    if (hipMemcpy(bins_host, bins_device, NUM_ELEMENTS * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
        printf("bins memcpy error from device to host\n");
    }

    //print_fractal(bins_host);
    printf("Elapsed kernel execution time: %f", aelapsedTime);

    // free GPU and host memory
    hipFree(bins_device);
    hipFree(seed_device);

    free(bins_host);
    free(seed_host);

    return 0;
}